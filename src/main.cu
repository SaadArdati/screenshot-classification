#include <stdio.h>
#include <stdlib.h>
#include <dirent.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include "cuda_utils.cuh"
#include "common.h"
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"

// Function declarations
extern "C" void extractFeaturesGPU(const unsigned char* h_images, int batch_size,
                                 int width, int height, int channels,
                                 Feature* h_features);

// Print device information
void printDeviceInfo() {
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDeviceProperties(&prop, 0));
    
    printf("\nCUDA Device Information:\n");
    printf("------------------------\n");
    printf("Device Name: %s\n", prop.name);
    printf("Compute Capability: %d.%d\n", prop.major, prop.minor);
    printf("Max Threads per Block: %d\n", prop.maxThreadsPerBlock);
    printf("Max Grid Dimensions: (%d, %d, %d)\n", 
           prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
    printf("Total Global Memory: %.2f GB\n", 
           (float)prop.totalGlobalMem / (1024.0f * 1024.0f * 1024.0f));
    printf("Shared Memory per Block: %zu KB\n", prop.sharedMemPerBlock / 1024);
    printf("\n");
}

// Load images in batches
int loadImageBatch(const char* dirpath, int label, unsigned char* batch_buffer,
                  Feature* features, int batch_size, int* current_index,
                  DIR* dir) {
    struct dirent* entry;
    int loaded = 0;
    int width, height, channels;

    while (loaded < batch_size && (entry = readdir(dir)) != NULL) {
        if (entry->d_type != DT_REG) continue;

        char fullpath[512];
        snprintf(fullpath, sizeof(fullpath), "%s/%s", dirpath, entry->d_name);

        // Load image
        unsigned char* img = stbi_load(fullpath, &width, &height, &channels, 3);
        if (!img) {
            fprintf(stderr, "Failed to load image: %s\n", fullpath);
            continue;
        }

        // Copy to batch buffer
        const int image_size = width * height * 3;
        memcpy(batch_buffer + (loaded * image_size), img, image_size);
        features[*current_index + loaded].label = label;

        stbi_image_free(img);
        loaded++;
    }

    return loaded;
}

int main(int argc, char** argv) {
    // Initialize CUDA
    CUDA_CHECK(hipSetDevice(0));
    printDeviceInfo();

    // Path to the split_data directory
    const char* screenshots_train_dir = "split_data/screenshots_256x256/train";
    const char* non_screenshots_train_dir = "split_data/non_screenshot_256x256/train";
    const char* model_path = (argc > 1) ? argv[1] : "trained_model.bin";

    // Allocate host memory for batch processing
    const int image_size = 256 * 256 * 3;  // Assuming 256x256 RGB images
    unsigned char* batch_buffer = (unsigned char*)malloc(MAX_BATCH_SIZE * image_size);
    Feature* features = (Feature*)malloc(MAX_BATCH_SIZE * sizeof(Feature));

    if (!batch_buffer || !features) {
        fprintf(stderr, "Failed to allocate host memory\n");
        return 1;
    }

    // Process screenshots
    DIR* dir = opendir(screenshots_train_dir);
    if (!dir) {
        fprintf(stderr, "Failed to open directory: %s\n", screenshots_train_dir);
        return 1;
    }

    int total_processed = 0;
    while (1) {
        int loaded = loadImageBatch(screenshots_train_dir, 1, batch_buffer,
                                  features, MAX_BATCH_SIZE, &total_processed, dir);
        if (loaded == 0) break;

        // Process batch on GPU
        extractFeaturesGPU(batch_buffer, loaded, 256, 256, 3, features + total_processed);
        total_processed += loaded;
    }
    closedir(dir);

    // Process non-screenshots (similar to screenshots)
    // ... (similar code for non-screenshots)

    // Clean up
    free(batch_buffer);
    free(features);

    return 0;
} 
#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <dirent.h>
#include <string.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include "cuda_utils.cuh"
#include "common.h"
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"

// Function declarations
extern "C" void extractFeaturesGPU(const unsigned char* h_images, int batch_size,
                                 int width, int height, int channels,
                                 Feature* h_features);
extern "C" void classifyBatchGPU(const Feature* train_features, int train_size,
                               const Feature* query_features, int query_size,
                               int* predictions, double* computation_times);

// Performance monitoring structure
typedef struct {
    double data_loading_time;
    double feature_extraction_time;
    double knn_transfer_time;
    double knn_compute_time;
    double total_time;
    size_t peak_memory_usage;
    int total_images;
    float accuracy;
} PerformanceMetrics;

// Print device information with power limits
void printDeviceInfo() {
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDeviceProperties(&prop, 0));
    
    printf("\nCUDA Device Information:\n");
    printf("------------------------\n");
    printf("Device Name: %s\n", prop.name);
    printf("Compute Capability: %d.%d\n", prop.major, prop.minor);
    printf("Max Threads per Block: %d\n", prop.maxThreadsPerBlock);
    printf("Max Grid Dimensions: (%d, %d, %d)\n", 
           prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
    printf("Total Global Memory: %.2f GB\n", 
           (float)prop.totalGlobalMem / (1024.0f * 1024.0f * 1024.0f));
    printf("Shared Memory per Block: %zu KB\n", prop.sharedMemPerBlock / 1024);
    printf("L2 Cache Size: %d KB\n", prop.l2CacheSize / 1024);
    printf("Max Threads per MultiProcessor: %d\n", prop.maxThreadsPerMultiProcessor);
    printf("Memory Clock Rate: %.2f GHz\n", prop.memoryClockRate * 1e-6);
    printf("Memory Bus Width: %d bits\n", prop.memoryBusWidth);
    printf("\n");
}

// Get current GPU memory usage
size_t getCurrentGPUMemory() {
    size_t free_mem, total_mem;
    CUDA_CHECK(hipMemGetInfo(&free_mem, &total_mem));
    return total_mem - free_mem;
}

// Print performance metrics
void printPerformanceMetrics(const PerformanceMetrics* metrics) {
    printf("\nDetailed Performance Metrics:\n");
    printf("---------------------------\n");
    printf("Data Loading Time: %.2f seconds\n", metrics->data_loading_time);
    printf("Feature Extraction Time: %.2f seconds\n", metrics->feature_extraction_time);
    printf("KNN Data Transfer Time: %.2f seconds\n", metrics->knn_transfer_time);
    printf("KNN Computation Time: %.2f seconds\n", metrics->knn_compute_time);
    printf("Total Processing Time: %.2f seconds\n", metrics->total_time);
    printf("Peak GPU Memory Usage: %.2f MB\n", metrics->peak_memory_usage / (1024.0f * 1024.0f));
    printf("Total Images Processed: %d\n", metrics->total_images);
    printf("Processing Speed: %.2f images/second\n", 
           metrics->total_images / metrics->total_time);
    printf("Classification Accuracy: %.2f%%\n", metrics->accuracy * 100);
    printf("\nPer-Phase Performance:\n");
    printf("Data Loading: %.1f%%\n", 
           (metrics->data_loading_time / metrics->total_time) * 100);
    printf("Feature Extraction: %.1f%%\n",
           (metrics->feature_extraction_time / metrics->total_time) * 100);
    printf("KNN Classification: %.1f%%\n",
           ((metrics->knn_transfer_time + metrics->knn_compute_time) / metrics->total_time) * 100);
}

// Load images in batches
int loadImageBatch(const char* dirpath, int label, unsigned char* batch_buffer,
                  Feature* features, int batch_size, int* current_index,
                  DIR* dir) {
    struct dirent* entry;
    int loaded = 0;
    int width, height, channels;

    while (loaded < batch_size && (entry = readdir(dir)) != NULL) {
        if (entry->d_type != DT_REG) continue;

        char fullpath[512];
        snprintf(fullpath, sizeof(fullpath), "%s/%s", dirpath, entry->d_name);

        // Load image
        unsigned char* img = stbi_load(fullpath, &width, &height, &channels, 3);
        if (!img) {
            fprintf(stderr, "Failed to load image: %s\n", fullpath);
            continue;
        }

        // Copy to batch buffer
        const int image_size = width * height * 3;
        memcpy(batch_buffer + (loaded * image_size), img, image_size);
        features[*current_index + loaded].label = label;

        stbi_image_free(img);
        loaded++;
    }

    return loaded;
}

// Save model to file
int saveModel(const char* filename, Feature* features, int total_features) {
    FILE* f = fopen(filename, "wb");
    if (!f) {
        fprintf(stderr, "Failed to open file for writing: %s\n", filename);
        return -1;
    }

    // Write number of features
    fwrite(&total_features, sizeof(int), 1, f);
    
    // Write features
    fwrite(features, sizeof(Feature), total_features, f);
    
    fclose(f);
    return 0;
}

// Update the main function to include KNN classification and performance monitoring
int main(int argc, char** argv) {
    PerformanceMetrics metrics = {0};
    size_t peak_memory = 0;
    clock_t total_start = clock();

    // Initialize CUDA and print device info
    CUDA_CHECK(hipSetDevice(0));
    printDeviceInfo();

    // Path to the split_data directory
    const char* screenshots_train_dir = "split_data/screenshots_256x256/train";
    const char* non_screenshots_train_dir = "split_data/non_screenshot_256x256/train";
    const char* model_path = (argc > 1) ? argv[1] : "trained_model.bin";

    // Performance measurement variables
    clock_t start_time, end_time;
    double total_time = 0.0, feature_extraction_time = 0.0;

    start_time = clock();

    // Allocate host memory for batch processing
    const int image_size = 256 * 256 * 3;  // Assuming 256x256 RGB images
    unsigned char* batch_buffer = (unsigned char*)malloc(MAX_BATCH_SIZE * image_size);
    Feature* all_features = (Feature*)malloc(100000 * sizeof(Feature)); // Adjust size as needed
    
    if (!batch_buffer || !all_features) {
        fprintf(stderr, "Failed to allocate host memory\n");
        return 1;
    }

    int total_processed = 0;

    // Process screenshots
    printf("Processing screenshots...\n");
    DIR* dir = opendir(screenshots_train_dir);
    if (!dir) {
        fprintf(stderr, "Failed to open directory: %s\n", screenshots_train_dir);
        return 1;
    }

    while (1) {
        int loaded = loadImageBatch(screenshots_train_dir, 1, batch_buffer,
                                  all_features, MAX_BATCH_SIZE, &total_processed, dir);
        if (loaded == 0) break;

        // Process batch on GPU
        clock_t batch_start = clock();
        extractFeaturesGPU(batch_buffer, loaded, 256, 256, 3, all_features + total_processed);
        feature_extraction_time += (double)(clock() - batch_start) / CLOCKS_PER_SEC;
        
        total_processed += loaded;
        printf("\rProcessed %d screenshots", total_processed);
        fflush(stdout);
    }
    printf("\n");
    closedir(dir);

    // Process non-screenshots
    printf("Processing non-screenshots...\n");
    dir = opendir(non_screenshots_train_dir);
    if (!dir) {
        fprintf(stderr, "Failed to open directory: %s\n", non_screenshots_train_dir);
        return 1;
    }

    while (1) {
        int loaded = loadImageBatch(non_screenshots_train_dir, 0, batch_buffer,
                                  all_features, MAX_BATCH_SIZE, &total_processed, dir);
        if (loaded == 0) break;

        // Process batch on GPU
        clock_t batch_start = clock();
        extractFeaturesGPU(batch_buffer, loaded, 256, 256, 3, all_features + total_processed);
        feature_extraction_time += (double)(clock() - batch_start) / CLOCKS_PER_SEC;

        total_processed += loaded;
        printf("\rProcessed %d non-screenshots", total_processed);
        fflush(stdout);
    }
    printf("\n");
    closedir(dir);

    // Save model
    printf("Saving model to %s...\n", model_path);
    if (saveModel(model_path, all_features, total_processed) != 0) {
        fprintf(stderr, "Failed to save model\n");
    }

    end_time = clock();
    total_time = (double)(end_time - start_time) / CLOCKS_PER_SEC;

    // Print performance metrics
    printf("\nPerformance Metrics:\n");
    printf("-------------------\n");
    printf("Total Processing Time: %.2f seconds\n", total_time);
    printf("Feature Extraction Time: %.2f seconds\n", feature_extraction_time);
    printf("Average Time per Image: %.4f seconds\n", feature_extraction_time / total_processed);
    printf("Total Images Processed: %d\n", total_processed);
    printf("Throughput: %.2f images/second\n", total_processed / total_time);

    // Record peak memory after feature extraction
    peak_memory = max(peak_memory, getCurrentGPUMemory());

    // Perform KNN classification on test set
    printf("\nPerforming KNN classification...\n");
    double knn_times[2];  // [0] = transfer time, [1] = compute time
    int* predictions = (int*)malloc(test_size * sizeof(int));
    
    clock_t knn_start = clock();
    classifyBatchGPU(train_features, train_size, test_features, test_size,
                     predictions, knn_times);
    clock_t knn_end = clock();

    // Record peak memory after KNN
    peak_memory = max(peak_memory, getCurrentGPUMemory());

    // Calculate accuracy
    int correct = 0;
    for (int i = 0; i < test_size; i++) {
        if (predictions[i] == test_features[i].label) {
            correct++;
        }
    }
    metrics.accuracy = (float)correct / test_size;

    // Update performance metrics
    metrics.data_loading_time = data_loading_time;
    metrics.feature_extraction_time = feature_extraction_time;
    metrics.knn_transfer_time = knn_times[0];
    metrics.knn_compute_time = knn_times[1];
    metrics.total_time = (double)(clock() - total_start) / CLOCKS_PER_SEC;
    metrics.peak_memory_usage = peak_memory;
    metrics.total_images = total_processed;

    // Print detailed performance metrics
    printPerformanceMetrics(&metrics);

    // Clean up
    free(batch_buffer);
    free(predictions);
    free(all_features);

    return 0;
} 
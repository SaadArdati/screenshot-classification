#include "hip/hip_runtime.h"
#ifdef _WIN32
#define NOMINMAX
#include <windows.h>
#endif

#include <stdio.h>
#include <stdlib.h>
#include <dirent.h>
#include <string.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <pthread.h>
#include <unistd.h>
#include "../include/cuda_utils.cuh"
#include "../include/common.h"
#define STB_IMAGE_IMPLEMENTATION
#include "../include/stb_image.h"

// Platform-specific directory handling
#ifdef _WIN32
#include <direct.h>
#define CREATE_DIR(dir) _mkdir(dir)
#define PATH_SEPARATOR "\\"
#else
#include <sys/stat.h>
#define CREATE_DIR(dir) mkdir(dir, 0777)
#define PATH_SEPARATOR "/"
#endif

// Function declarations
extern "C" void extractFeaturesGPU(const unsigned char* h_images, int batch_size,
                                 int width, int height, int channels,
                                 Feature* h_features);
extern "C" void classifyBatchGPU(const Feature* train_features, int train_size,
                               const Feature* query_features, int query_size,
                               int* predictions, double* computation_times);

// Performance monitoring structure
typedef struct {
    double data_loading_time;
    double feature_extraction_time;
    double knn_transfer_time;
    double knn_compute_time;
    double total_time;
    size_t peak_memory_usage;
    int total_images;
    float accuracy;
    int statistical_detections; // Added for tracking statistical detections
} PerformanceMetrics;

// Asynchronous loading structure
typedef struct {
    char dirpath[512];
    int label;
    unsigned char* batch_buffer;
    Feature* features;
    int batch_size;
    int* current_index;
    DIR* dir;
    int loaded;
    int done;
    pthread_mutex_t mutex;
} LoadBatchArgs;

// Print device information with power limits
void printDeviceInfo() {
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDeviceProperties(&prop, 0));
    
    printf("\nCUDA Device Information:\n");
    printf("------------------------\n");
    printf("Device Name: %s\n", prop.name);
    printf("Compute Capability: %d.%d\n", prop.major, prop.minor);
    printf("Max Threads per Block: %d\n", prop.maxThreadsPerBlock);
    printf("Max Grid Dimensions: (%d, %d, %d)\n", 
           prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
    printf("Total Global Memory: %.2f GB\n", 
           (float)prop.totalGlobalMem / (1024.0f * 1024.0f * 1024.0f));
    printf("Shared Memory per Block: %d KB\n", (int)(prop.sharedMemPerBlock / 1024));
    printf("L2 Cache Size: %d KB\n", (int)(prop.l2CacheSize / 1024));
    printf("Max Threads per MultiProcessor: %d\n", prop.maxThreadsPerMultiProcessor);
    printf("Memory Clock Rate: %.2f GHz\n", prop.memoryClockRate / (1000.0f * 1000.0f));
    printf("Memory Bus Width: %d bits\n", prop.memoryBusWidth);
    printf("\n");
}

// Get current GPU memory usage
size_t getCurrentGPUMemory() {
    size_t free_mem, total_mem;
    CUDA_CHECK(hipMemGetInfo(&free_mem, &total_mem));
    return total_mem - free_mem;
}

// Save model to file
int saveModel(const char* path, const Feature* features, int count) {
    FILE* f = fopen(path, "wb");
    if (!f) return -1;
    
    fwrite(&count, sizeof(int), 1, f);
    fwrite(features, sizeof(Feature), count, f);
    fclose(f);
    return 0;
}

// Print performance metrics
void printPerformanceMetrics(const PerformanceMetrics* metrics) {
    printf("\nDetailed Performance Metrics:\n");
    printf("---------------------------\n");
    printf("Data Loading Time: %.2f seconds\n", metrics->data_loading_time);
    printf("Feature Extraction Time: %.2f seconds\n", metrics->feature_extraction_time);
    printf("KNN Data Transfer Time: %.2f seconds\n", metrics->knn_transfer_time);
    printf("KNN Computation Time: %.2f seconds\n", metrics->knn_compute_time);
    printf("Total Processing Time: %.2f seconds\n", metrics->total_time);
    printf("Peak GPU Memory Usage: %.2f MB\n", metrics->peak_memory_usage / (1024.0f * 1024.0f));
    printf("Total Images Processed: %d\n", metrics->total_images);
    printf("Images Classified by Statistical Analysis: %d (%.1f%%)\n", 
           metrics->statistical_detections, 
           (float)metrics->statistical_detections / metrics->total_images * 100);
    printf("Processing Speed: %.2f images/second\n", 
           metrics->total_images / metrics->total_time);
    printf("Classification Accuracy: %.2f%%\n", metrics->accuracy * 100);
    printf("\nPer-Phase Performance:\n");
    printf("Data Loading: %.1f%%\n", 
           (metrics->data_loading_time / metrics->total_time) * 100);
    printf("Feature Extraction: %.1f%%\n",
           (metrics->feature_extraction_time / metrics->total_time) * 100);
    printf("KNN Classification: %.1f%%\n",
           ((metrics->knn_transfer_time + metrics->knn_compute_time) / metrics->total_time) * 100);
}

// Asynchronous batch loading function
void* loadBatchAsync(void* args) {
    LoadBatchArgs* load_args = (LoadBatchArgs*)args;
    struct dirent* entry;
    int width, height, channels;
    int loaded = 0;
    
    pthread_mutex_lock(&load_args->mutex);
    DIR* dir = load_args->dir;
    pthread_mutex_unlock(&load_args->mutex);
    
    while (loaded < load_args->batch_size && (entry = readdir(dir)) != NULL) {
        if (entry->d_type != DT_REG) continue;
        
        char fullpath[512];
        #ifdef _WIN32
        snprintf(fullpath, sizeof(fullpath), "%s%s%s", load_args->dirpath, PATH_SEPARATOR, entry->d_name);
        #else
        snprintf(fullpath, sizeof(fullpath), "%s/%s", load_args->dirpath, entry->d_name);
        #endif
        
        // Load image
        unsigned char* img = stbi_load(fullpath, &width, &height, &channels, 3);
        if (!img) {
            fprintf(stderr, "Failed to load image: %s\n", fullpath);
            continue;
        }
        
        // Copy to batch buffer
        const int image_size = width * height * 3;
        pthread_mutex_lock(&load_args->mutex);
        memcpy(load_args->batch_buffer + (loaded * image_size), img, image_size);
        load_args->features[*load_args->current_index + loaded].label = load_args->label;
        pthread_mutex_unlock(&load_args->mutex);
        
        stbi_image_free(img);
        loaded++;
    }
    
    pthread_mutex_lock(&load_args->mutex);
    load_args->loaded = loaded;
    load_args->done = 1;
    pthread_mutex_unlock(&load_args->mutex);
    
    return NULL;
}

// Load a batch of images asynchronously
int startLoadBatch(LoadBatchArgs* args, pthread_t* thread) {
    args->loaded = 0;
    args->done = 0;
    pthread_mutex_init(&args->mutex, NULL);
    
    if (pthread_create(thread, NULL, loadBatchAsync, args) != 0) {
        fprintf(stderr, "Failed to create loading thread\n");
        return 0;
    }
    
    return 1;
}

// Wait for batch loading to complete
int waitForBatchLoading(LoadBatchArgs* args, pthread_t thread) {
    pthread_join(thread, NULL);
    pthread_mutex_destroy(&args->mutex);
    return args->loaded;
}

// Count statistical detections in a feature batch
int countStatisticalDetections(Feature* features, int count) {
    int detections = 0;
    for (int i = 0; i < count; i++) {
        if (features[i].label == 2) { // Special code for statistical detection
            detections++;
        }
    }
    return detections;
}

// Load all images from a directory with batching
int load_folder(const char *dirpath, const int label, Feature **arr, int *size) {
    DIR *d = opendir(dirpath);
    if (!d) {
        fprintf(stderr, "Error opening directory: %s\n", dirpath);
        return -1;
    }
    
    // Count files first
    struct dirent *entry;
    int file_count = 0;
    while ((entry = readdir(d)) != NULL) {
        if (entry->d_type == DT_REG) {
            file_count++;
        }
    }
    rewinddir(d);
    
    printf("Found %d files in %s\n", file_count, dirpath);
    
    // Allocate memory for file paths
    char **filepaths = (char**)malloc(file_count * sizeof(char*));
    if (!filepaths) {
        fprintf(stderr, "Memory allocation failed\n");
        closedir(d);
        return -1;
    }
    
    // Store file paths
    int count = 0;
    while ((entry = readdir(d)) != NULL && count < file_count) {
        if (entry->d_type == DT_REG) {
            filepaths[count] = (char*)malloc(512 * sizeof(char));
            if (!filepaths[count]) {
                fprintf(stderr, "Memory allocation failed\n");
                for (int i = 0; i < count; i++) {
                    free(filepaths[i]);
                }
                free(filepaths);
                closedir(d);
                return -1;
            }
            snprintf(filepaths[count], 512, "%s/%s", dirpath, entry->d_name);
            count++;
        }
    }
    closedir(d);
    
    // Process in batches
    const int batch_size = MAX_BATCH_SIZE;
    unsigned char *batch_images = NULL;
    Feature *batch_features = NULL;
    int width = 0, height = 0;
    
    for (int batch_start = 0; batch_start < count; batch_start += batch_size) {
        int current_batch_size = (batch_start + batch_size > count) ? 
                               (count - batch_start) : batch_size;
        
        printf("Processing batch %d/%d with %d images...\n", 
               batch_start / batch_size + 1, 
               (count + batch_size - 1) / batch_size, 
               current_batch_size);
        
        // Load batch images
        for (int i = 0; i < current_batch_size; i++) {
            int img_width, img_height, channels;
            unsigned char *img = stbi_load(filepaths[batch_start + i], 
                                         &img_width, &img_height, &channels, 3);
            
            if (!img) {
                fprintf(stderr, "Failed to load image: %s\n", filepaths[batch_start + i]);
                continue;
            }
            
            // Initialize batch arrays on first image
            if (batch_images == NULL) {
                width = img_width;
                height = img_height;
                batch_images = (unsigned char*)malloc(batch_size * width * height * 3 * sizeof(unsigned char));
                batch_features = (Feature*)malloc(batch_size * sizeof(Feature));
                
                if (!batch_images || !batch_features) {
                    fprintf(stderr, "Memory allocation failed\n");
                    stbi_image_free(img);
                    free(batch_images);
                    free(batch_features);
                    for (int j = 0; j < count; j++) {
                        free(filepaths[j]);
                    }
                    free(filepaths);
                    return -1;
                }
            }
            
            // Check if image dimensions match
            if (img_width != width || img_height != height) {
                fprintf(stderr, "Image dimensions do not match: %s\n", filepaths[batch_start + i]);
                stbi_image_free(img);
                continue;
            }
            
            // Copy image data to batch buffer
            memcpy(batch_images + (i * width * height * 3), img, width * height * 3 * sizeof(unsigned char));
            stbi_image_free(img);
        }
        
        // Extract features using GPU
        extractFeaturesGPU(batch_images, current_batch_size, width, height, 3, batch_features);
        
        // Set labels
        for (int i = 0; i < current_batch_size; i++) {
            batch_features[i].label = label;
        }
        
        // Append features to result array
        int new_size = *size + current_batch_size;
        *arr = (Feature*)realloc(*arr, new_size * sizeof(Feature));
        if (!*arr) {
            fprintf(stderr, "Memory allocation failed\n");
            free(batch_images);
            free(batch_features);
            for (int j = 0; j < count; j++) {
                free(filepaths[j]);
            }
            free(filepaths);
            return -1;
        }
        
        memcpy(*arr + *size, batch_features, current_batch_size * sizeof(Feature));
        *size = new_size;
    }
    
    // Clean up
    free(batch_images);
    free(batch_features);
    for (int i = 0; i < count; i++) {
        free(filepaths[i]);
    }
    free(filepaths);
    
    return 0;
}

// Save model to file
int save_model(const char *filename, Feature *dataset, int size) {
    FILE *f = fopen(filename, "wb");
    if (!f) {
        fprintf(stderr, "Failed to open file for writing: %s\n", filename);
        return -1;
    }
    
    fwrite(&size, sizeof(int), 1, f);
    fwrite(dataset, sizeof(Feature), size, f);
    fclose(f);
    
    printf("Model saved to %s with %d features\n", filename, size);
    return 0;
}

// Evaluate model using cross-validation
float evaluate_model(Feature *train_set, int train_size, Feature *test_set, int test_size) {
    printf("Evaluating model on %d test samples...\n", test_size);
    
    int *predictions = (int*)malloc(test_size * sizeof(int));
    int correct = 0;
    
    // Process test set in batches
    double computation_times[2] = {0, 0};
    classifyBatchGPU(train_set, train_size, test_set, test_size, predictions, computation_times);
    
    // Calculate accuracy
    for (int i = 0; i < test_size; i++) {
        if (predictions[i] == test_set[i].label) {
            correct++;
        }
    }
    
    float accuracy = (float)correct / test_size;
    printf("Accuracy: %.2f%% (%d/%d)\n", accuracy * 100, correct, test_size);
    
    free(predictions);
    return accuracy;
}

// Update the main function to include KNN classification and performance monitoring
int main(int argc, char** argv) {
    PerformanceMetrics metrics = {0};
    size_t peak_memory = 0;
    clock_t total_start = clock();
    double data_loading_time = 0.0;
    double feature_extraction_time = 0.0;
    int statistical_detections = 0;

    // Initialize CUDA and print device info
    CUDA_CHECK(hipSetDevice(0));
    printDeviceInfo();

    // Get current working directory
    char cwd[1024];
    if (getcwd(cwd, sizeof(cwd)) == NULL) {
        fprintf(stderr, "Failed to get current working directory\n");
        return 1;
    }
    
    // Path to the split_data directory with platform-specific separators
    char screenshots_train_dir[1024], non_screenshots_train_dir[1024];
    char screenshots_test_dir[1024], non_screenshots_test_dir[1024];
    char model_path[1024];
    
    #ifdef _WIN32
    snprintf(screenshots_train_dir, sizeof(screenshots_train_dir), 
             "%s%s%s%s%s%s%s", cwd, PATH_SEPARATOR, "..", PATH_SEPARATOR, "split_data", PATH_SEPARATOR, "screenshots_256x256", PATH_SEPARATOR, "train");
    snprintf(non_screenshots_train_dir, sizeof(non_screenshots_train_dir), 
             "%s%s%s%s%s%s%s", cwd, PATH_SEPARATOR, "..", PATH_SEPARATOR, "split_data", PATH_SEPARATOR, "non_screenshot_256x256", PATH_SEPARATOR, "train");
    snprintf(screenshots_test_dir, sizeof(screenshots_test_dir), 
             "%s%s%s%s%s%s%s", cwd, PATH_SEPARATOR, "..", PATH_SEPARATOR, "split_data", PATH_SEPARATOR, "screenshots_256x256", PATH_SEPARATOR, "test");
    snprintf(non_screenshots_test_dir, sizeof(non_screenshots_test_dir), 
             "%s%s%s%s%s%s%s", cwd, PATH_SEPARATOR, "..", PATH_SEPARATOR, "split_data", PATH_SEPARATOR, "non_screenshot_256x256", PATH_SEPARATOR, "test");
    snprintf(model_path, sizeof(model_path), "%s%s%s", cwd, PATH_SEPARATOR, "trained_model.bin");
    #else
    snprintf(screenshots_train_dir, sizeof(screenshots_train_dir), 
             "%s/split_data/screenshots_256x256/train", cwd);
    snprintf(non_screenshots_train_dir, sizeof(non_screenshots_train_dir), 
             "%s/split_data/non_screenshot_256x256/train", cwd);
    snprintf(screenshots_test_dir, sizeof(screenshots_test_dir), 
             "%s/split_data/screenshots_256x256/test", cwd);
    snprintf(non_screenshots_test_dir, sizeof(non_screenshots_test_dir), 
             "%s/split_data/non_screenshot_256x256/test", cwd);
    snprintf(model_path, sizeof(model_path), "%s/trained_model.bin", cwd);
    #endif
    
    printf("Using directories:\n");
    printf("Screenshots train: %s\n", screenshots_train_dir);
    printf("Non-screenshots train: %s\n", non_screenshots_train_dir);
    printf("Screenshots test: %s\n", screenshots_test_dir);
    printf("Non-screenshots test: %s\n", non_screenshots_test_dir);
    printf("Model path: %s\n\n", model_path);
    
    // Parse command line arguments
    if (argc > 1) {
        strncpy(model_path, argv[1], sizeof(model_path) - 1);
    }
    
    // Allocate memory for features and batch processing
    Feature* all_features = (Feature*)malloc(MAX_IMAGES * sizeof(Feature));
    Feature* test_features = (Feature*)malloc(MAX_IMAGES * sizeof(Feature));
    unsigned char* batch_buffer = (unsigned char*)malloc(MAX_BATCH_SIZE * 256 * 256 * 3);
    unsigned char* next_batch_buffer = (unsigned char*)malloc(MAX_BATCH_SIZE * 256 * 256 * 3);
    
    if (!all_features || !test_features || !batch_buffer || !next_batch_buffer) {
        fprintf(stderr, "Failed to allocate memory\n");
        return 1;
    }
    
    // Initialize counters
    int train_size = 0;
    int test_size = 0;
    
    // Start loading data
    clock_t load_start = clock();
    
    // Process training screenshots
    printf("Processing training screenshots...\n");
    DIR* dir = opendir(screenshots_train_dir);
    if (!dir) {
        fprintf(stderr, "Failed to open directory: %s\n", screenshots_train_dir);
        return 1;
    }
    
    // Set up asynchronous loading
    LoadBatchArgs load_args = {0};
    strncpy(load_args.dirpath, screenshots_train_dir, sizeof(load_args.dirpath) - 1);
    load_args.label = 1;
    load_args.batch_buffer = batch_buffer;
    load_args.features = all_features;
    load_args.batch_size = MAX_BATCH_SIZE;
    load_args.current_index = &train_size;
    load_args.dir = dir;
    
    pthread_t loading_thread;
    if (!startLoadBatch(&load_args, &loading_thread)) {
        closedir(dir);
        return 1;
    }
    
    // Process batches with overlapped loading and computation
    while (1) {
        // Wait for current batch to load
        int loaded = waitForBatchLoading(&load_args, loading_thread);
        if (loaded == 0) break;
        
        // Start loading next batch
        LoadBatchArgs next_load_args = load_args;
        next_load_args.batch_buffer = next_batch_buffer;
        pthread_t next_loading_thread;
        int next_batch_loading = startLoadBatch(&next_load_args, &next_loading_thread);
        
        // Process current batch on GPU
        clock_t batch_start = clock();
        extractFeaturesGPU(batch_buffer, loaded, 256, 256, 3, all_features + train_size);
        feature_extraction_time += (double)(clock() - batch_start) / CLOCKS_PER_SEC;
        
        // Count statistical detections
        statistical_detections += countStatisticalDetections(all_features + train_size, loaded);
        
        train_size += loaded;
        printf("\rProcessed %d training screenshots", train_size);
        fflush(stdout);
        
        // Swap buffers for next iteration
        unsigned char* temp = batch_buffer;
        batch_buffer = next_batch_buffer;
        next_batch_buffer = temp;
        
        // Update load args for next iteration
        load_args = next_load_args;
        loading_thread = next_loading_thread;
        
        if (!next_batch_loading) break;
    }
    printf("\n");
    closedir(dir);
    
    // Process training non-screenshots
    printf("Processing training non-screenshots...\n");
    dir = opendir(non_screenshots_train_dir);
    if (!dir) {
        fprintf(stderr, "Failed to open directory: %s\n", non_screenshots_train_dir);
        return 1;
    }
    
    int total_processed = train_size;
    
    // Set up asynchronous loading for non-screenshots
    strncpy(load_args.dirpath, non_screenshots_train_dir, sizeof(load_args.dirpath) - 1);
    load_args.label = 0;
    load_args.batch_buffer = batch_buffer;
    load_args.features = all_features;
    load_args.batch_size = MAX_BATCH_SIZE;
    load_args.current_index = &train_size;
    load_args.dir = dir;
    
    if (!startLoadBatch(&load_args, &loading_thread)) {
        closedir(dir);
        return 1;
    }
    
    // Process batches with overlapped loading and computation
    while (1) {
        // Wait for current batch to load
        int loaded = waitForBatchLoading(&load_args, loading_thread);
        if (loaded == 0) break;
        
        // Start loading next batch
        LoadBatchArgs next_load_args = load_args;
        next_load_args.batch_buffer = next_batch_buffer;
        pthread_t next_loading_thread;
        int next_batch_loading = startLoadBatch(&next_load_args, &next_loading_thread);
        
        // Process current batch on GPU
        clock_t batch_start = clock();
        extractFeaturesGPU(batch_buffer, loaded, 256, 256, 3, all_features + train_size);
        feature_extraction_time += (double)(clock() - batch_start) / CLOCKS_PER_SEC;
        
        // Count statistical detections (should be 0 for non-screenshots ideally)
        statistical_detections += countStatisticalDetections(all_features + train_size, loaded);
        
        train_size += loaded;
        printf("\rProcessed %d training non-screenshots", train_size - total_processed);
        fflush(stdout);
        
        // Swap buffers for next iteration
        unsigned char* temp = batch_buffer;
        batch_buffer = next_batch_buffer;
        next_batch_buffer = temp;
        
        // Update load args for next iteration
        load_args = next_load_args;
        loading_thread = next_loading_thread;
        
        if (!next_batch_loading) break;
    }
    printf("\n");
    closedir(dir);
    
    // Process test data
    printf("Processing test data...\n");
    
    // Test screenshots
    dir = opendir(screenshots_test_dir);
    if (dir) {
        // Set up asynchronous loading for test screenshots
        strncpy(load_args.dirpath, screenshots_test_dir, sizeof(load_args.dirpath) - 1);
        load_args.label = 1;
        load_args.batch_buffer = batch_buffer;
        load_args.features = test_features;
        load_args.batch_size = MAX_BATCH_SIZE;
        load_args.current_index = &test_size;
        load_args.dir = dir;
        
        if (!startLoadBatch(&load_args, &loading_thread)) {
            closedir(dir);
            return 1;
        }
        
        // Process batches with overlapped loading and computation
        while (1) {
            // Wait for current batch to load
            int loaded = waitForBatchLoading(&load_args, loading_thread);
            if (loaded == 0) break;
            
            // Start loading next batch
            LoadBatchArgs next_load_args = load_args;
            next_load_args.batch_buffer = next_batch_buffer;
            pthread_t next_loading_thread;
            int next_batch_loading = startLoadBatch(&next_load_args, &next_loading_thread);
            
            // Process current batch on GPU
            clock_t batch_start = clock();
            extractFeaturesGPU(batch_buffer, loaded, 256, 256, 3, test_features + test_size);
            feature_extraction_time += (double)(clock() - batch_start) / CLOCKS_PER_SEC;
            
            // Count statistical detections in test data
            statistical_detections += countStatisticalDetections(test_features + test_size, loaded);
            
            test_size += loaded;
            printf("\rProcessed %d test screenshots", test_size);
            fflush(stdout);
            
            // Swap buffers for next iteration
            unsigned char* temp = batch_buffer;
            batch_buffer = next_batch_buffer;
            next_batch_buffer = temp;
            
            // Update load args for next iteration
            load_args = next_load_args;
            loading_thread = next_loading_thread;
            
            if (!next_batch_loading) break;
        }
        printf("\n");
        closedir(dir);
    }
    
    // Test non-screenshots
    dir = opendir(non_screenshots_test_dir);
    if (dir) {
        // Set up asynchronous loading for test non-screenshots
        strncpy(load_args.dirpath, non_screenshots_test_dir, sizeof(load_args.dirpath) - 1);
        load_args.label = 0;
        load_args.batch_buffer = batch_buffer;
        load_args.features = test_features;
        load_args.batch_size = MAX_BATCH_SIZE;
        load_args.current_index = &test_size;
        load_args.dir = dir;
        
        if (!startLoadBatch(&load_args, &loading_thread)) {
            closedir(dir);
            return 1;
        }
        
        // Process batches with overlapped loading and computation
        while (1) {
            // Wait for current batch to load
            int loaded = waitForBatchLoading(&load_args, loading_thread);
            if (loaded == 0) break;
            
            // Start loading next batch
            LoadBatchArgs next_load_args = load_args;
            next_load_args.batch_buffer = next_batch_buffer;
            pthread_t next_loading_thread;
            int next_batch_loading = startLoadBatch(&next_load_args, &next_loading_thread);
            
            // Process current batch on GPU
            clock_t batch_start = clock();
            extractFeaturesGPU(batch_buffer, loaded, 256, 256, 3, test_features + test_size);
            feature_extraction_time += (double)(clock() - batch_start) / CLOCKS_PER_SEC;
            
            // Count statistical detections in test data (should be minimal for non-screenshots)
            statistical_detections += countStatisticalDetections(test_features + test_size, loaded);
            
            test_size += loaded;
            printf("\rProcessed %d test non-screenshots", test_size);
            fflush(stdout);
            
            // Swap buffers for next iteration
            unsigned char* temp = batch_buffer;
            batch_buffer = next_batch_buffer;
            next_batch_buffer = temp;
            
            // Update load args for next iteration
            load_args = next_load_args;
            loading_thread = next_loading_thread;
            
            if (!next_batch_loading) break;
        }
        printf("\n");
        closedir(dir);
    }
    data_loading_time = (double)(clock() - load_start) / CLOCKS_PER_SEC;
    
    // Convert label=2 (statistical detection) to label=1 (screenshot) for saving
    for (int i = 0; i < train_size; i++) {
        if (all_features[i].label == 2) all_features[i].label = 1;
    }
    
    // Save model
    printf("Saving model to %s...\n", model_path);
    if (saveModel(model_path, all_features, train_size) != 0) {
        fprintf(stderr, "Failed to save model\n");
    }
    
    // Record peak memory after feature extraction
    peak_memory = max(peak_memory, getCurrentGPUMemory());
    
    // Perform KNN classification on test set (only for examples not classified by statistical analysis)
    printf("\nPerforming KNN classification...\n");
    double knn_times[2] = {0};  // [0] = transfer time, [1] = compute time
    int* predictions = (int*)malloc(test_size * sizeof(int));
    
    if (!predictions) {
        fprintf(stderr, "Failed to allocate memory for predictions\n");
        return 1;
    }
    
    classifyBatchGPU(all_features, train_size, test_features, test_size, predictions, knn_times);
    
    // Calculate accuracy - consider both KNN and statistical detections
    int correct = 0;
    for (int i = 0; i < test_size; i++) {
        // For statistical detections (label=2), the prediction is always 1 (screenshot)
        int actual_label = (test_features[i].label == 2) ? 1 : test_features[i].label;
        if (predictions[i] == actual_label) {
            correct++;
        }
    }
    metrics.accuracy = (float)correct / test_size;
    
    // Update performance metrics
    metrics.data_loading_time = data_loading_time;
    metrics.feature_extraction_time = feature_extraction_time;
    metrics.knn_transfer_time = knn_times[0];
    metrics.knn_compute_time = knn_times[1];
    metrics.total_time = (double)(clock() - total_start) / CLOCKS_PER_SEC;
    metrics.peak_memory_usage = peak_memory;
    metrics.total_images = train_size + test_size;
    metrics.statistical_detections = statistical_detections;
    
    // Print detailed performance metrics
    printPerformanceMetrics(&metrics);
    
    // Clean up
    free(batch_buffer);
    free(next_batch_buffer);
    free(predictions);
    free(all_features);
    free(test_features);
    
    return 0;
}

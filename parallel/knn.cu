#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "../include/cuda_utils.cuh"
#include "../include/common.h"

// Structure for storing distance and label pairs
typedef struct {
    float distance;
    int label;
} DistanceLabel;

// CUDA kernel for computing weighted distances between one query and all training examples
__global__ void computeDistancesKernel(
    const Feature* train_features,
    const Feature* query_feature,
    DistanceLabel* distances,
    int train_size
) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= train_size) return;

    // Define weights for different feature components
    const float hist_weight = 0.3f;
    const float edge_weight = 0.3f;
    const float top_weight = 0.2f;
    const float bottom_weight = 0.2f;
    
    float sum = 0.0f;
    
    // Regular histogram distance
    #pragma unroll
    for (int i = 0; i < NUM_BINS; i++) {
        const float diff = train_features[idx].bins[i] - query_feature->bins[i];
        sum += hist_weight * diff * diff;
    }

    // Edge histogram distance
    #pragma unroll
    for (int i = 0; i < NUM_BINS; i++) {
        const float diff = train_features[idx].edge_bins[i] - query_feature->edge_bins[i];
        sum += edge_weight * diff * diff;
    }
    
    // Top region histogram distance
    #pragma unroll
    for (int i = 0; i < NUM_BINS; i++) {
        const float diff = train_features[idx].top_region_bins[i] - query_feature->top_region_bins[i];
        sum += top_weight * diff * diff;
    }
    
    // Bottom region histogram distance
    #pragma unroll
    for (int i = 0; i < NUM_BINS; i++) {
        const float diff = train_features[idx].bottom_region_bins[i] - query_feature->bottom_region_bins[i];
        sum += bottom_weight * diff * diff;
    }

    distances[idx].distance = sqrtf(sum);
    distances[idx].label = train_features[idx].label;
}

// CUDA kernel for parallel reduction to find K nearest neighbors
__global__ void findTopKKernel(
    DistanceLabel* distances,
    int n,
    DistanceLabel* top_k,
    int k
) {
    extern __shared__ DistanceLabel shared_distances[];

    const int tid = threadIdx.x;
    const int gid = blockIdx.x * blockDim.x + threadIdx.x;

    // Initialize shared memory with maximum values
    if (tid < k) {
        shared_distances[tid].distance = INFINITY;
        shared_distances[tid].label = -1;
    }
    __syncthreads();

    // Each thread processes one element
    if (gid < n) {
        // Insert into local top-k if distance is smaller
        for (int i = 0; i < k; i++) {
            if (distances[gid].distance < shared_distances[i].distance) {
                // Shift elements to make room
                for (int j = k-1; j > i; j--) {
                    shared_distances[j] = shared_distances[j-1];
                }
                
                // Insert new element
                shared_distances[i] = distances[gid];
                break;
            }
        }
    }

    __syncthreads();

    // First thread writes results to global memory
    if (tid == 0) {
        for (int i = 0; i < k; i++) {
            top_k[blockIdx.x * k + i] = shared_distances[i];
        }
    }
}

// CUDA kernel for majority voting
__global__ void majorityVoteKernel(
    DistanceLabel* top_k,
    int* predictions,
    int batch_size,
    int k
) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= batch_size) return;

    // Count votes for each class (0 or 1)
    int votes_for_screenshot = 0;
    for (int i = 0; i < k; i++) {
        if (top_k[idx * k + i].label == 1) {
            votes_for_screenshot++;
        }
    }

    // Majority vote threshold should match CPU implementation
    predictions[idx] = (votes_for_screenshot >= (k / 2 + 1)) ? 1 : 0;
}

// Host function to classify a batch of query features
extern "C" void classifyBatchGPU(
    const Feature* train_features,
    int train_size,
    const Feature* query_features,
    int query_size,
    int* predictions,
    double* computation_times
) {
    // Skip queries that were already classified by statistical analysis
    int actual_query_size = 0;
    int* query_indices = (int*)malloc(query_size * sizeof(int));
    
    for (int i = 0; i < query_size; i++) {
        if (query_features[i].label != 2) {  // Not statistically detected
            query_indices[actual_query_size++] = i;
        } else {
            // For queries already classified as screenshots by statistical analysis
            predictions[i] = 1;  // Mark as screenshot
        }
    }
    
    // If all queries were already classified, no need for KNN
    if (actual_query_size == 0) {
        computation_times[0] = 0;
        computation_times[1] = 0;
        free(query_indices);
        return;
    }
    
    // Create temporary array for actual queries
    Feature* actual_queries = (Feature*)malloc(actual_query_size * sizeof(Feature));
    int* actual_predictions = (int*)malloc(actual_query_size * sizeof(int));
    
    for (int i = 0; i < actual_query_size; i++) {
        actual_queries[i] = query_features[query_indices[i]];
    }
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    // Allocate device memory
    Feature* d_train_features = allocateDeviceMemory<Feature>(train_size);
    Feature* d_query_features = allocateDeviceMemory<Feature>(actual_query_size);
    DistanceLabel* d_distances = allocateDeviceMemory<DistanceLabel>(train_size * actual_query_size);
    DistanceLabel* d_top_k = allocateDeviceMemory<DistanceLabel>(actual_query_size * K_NEIGHBORS);
    int* d_predictions = allocateDeviceMemory<int>(actual_query_size);
    
    // Copy training data to device (only once)
    hipEventRecord(start);
    copyToDevice(d_train_features, train_features, train_size);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float transfer_time;
    hipEventElapsedTime(&transfer_time, start, stop);
    computation_times[0] = transfer_time / 1000.0; // Convert to seconds
    
    // Copy query features to device
    hipEventRecord(start);
    copyToDevice(d_query_features, actual_queries, actual_query_size);
    
    // Calculate grid and block dimensions
    const int block_size = THREADS_PER_BLOCK;
    const int num_blocks_distance = (train_size + block_size - 1) / block_size;
    const int num_blocks_query = (actual_query_size + block_size - 1) / block_size;
    
    // Process each query
    for (int i = 0; i < actual_query_size; i++) {
        // Compute distances
        computeDistancesKernel<<<num_blocks_distance, block_size>>>(
            d_train_features,
            &d_query_features[i],
            &d_distances[i * train_size],
            train_size
        );
        CUDA_CHECK_KERNEL();
        
        // Find top-K nearest neighbors
        findTopKKernel<<<num_blocks_query, block_size, K_NEIGHBORS * sizeof(DistanceLabel)>>>(
            &d_distances[i * train_size],
            train_size,
            &d_top_k[i * K_NEIGHBORS],
            K_NEIGHBORS
        );
        CUDA_CHECK_KERNEL();
    }
    
    // Perform majority voting
    majorityVoteKernel<<<num_blocks_query, block_size>>>(
        d_top_k,
        d_predictions,
        actual_query_size,
        K_NEIGHBORS
    );
    CUDA_CHECK_KERNEL();
    
    // Copy results back to host
    copyToHost(actual_predictions, d_predictions, actual_query_size);
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float compute_time;
    hipEventElapsedTime(&compute_time, start, stop);
    computation_times[1] = compute_time / 1000.0; // Convert to seconds
    
    // Map results back to original indices
    for (int i = 0; i < actual_query_size; i++) {
        predictions[query_indices[i]] = actual_predictions[i];
    }
    
    // Clean up
    freeDeviceMemory(d_train_features);
    freeDeviceMemory(d_query_features);
    freeDeviceMemory(d_distances);
    freeDeviceMemory(d_top_k);
    freeDeviceMemory(d_predictions);
    
    free(query_indices);
    free(actual_queries);
    free(actual_predictions);
    
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

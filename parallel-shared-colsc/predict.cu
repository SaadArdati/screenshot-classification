#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include "../include/cuda_utils.cuh"
#include "../include/common.h"
#include "../include/screenshot_utils.h"
#define STB_IMAGE_IMPLEMENTATION
#include "../include/stb_image.h"

// Function declarations - these are implemented in feature_extraction.cu and knn.cu
extern "C" void extractFeaturesGPU(const unsigned char* h_images, int batch_size,
                                  int width, int height, int channels, 
                                  Feature* h_features);
extern "C" void classifyBatchGPU(const Feature* train_features, int train_size,
                                const Feature* query_features, int query_size,
                                int* predictions, double* computation_times);

// Constants in device constant memory for faster access
__constant__ int c_edge_threshold = EDGE_THRESHOLD;

// Helper struct for thread-local counters to reduce atomic operations
typedef struct {
    int edge_pixels;
    int regular_edge_pixels;
    int uniform_color_pixels;
} PixelCounters;

// CUDA kernel for computing screenshot statistics with optimized memory access
__global__ void computeScreenshotStatsKernel(
    const unsigned char* d_img, 
    int w, int h, int channels,
    int* d_edge_pixels,
    int* d_regular_edge_pixels,
    int* d_uniform_color_pixels,
    int* d_horizontal_edge_counts) {
    
    // Thread block-level shared memory for counter accumulation
    __shared__ int s_edge_pixels;
    __shared__ int s_regular_edge_pixels;
    __shared__ int s_uniform_color_pixels;
    
    // Initialize shared memory
    if (threadIdx.x == 0 && threadIdx.y == 0) {
        s_edge_pixels = 0;
        s_regular_edge_pixels = 0;
        s_uniform_color_pixels = 0;
    }
    __syncthreads();
    
    // Thread-local counters to reduce atomic operations
    PixelCounters local = {0, 0, 0};
    
    // Calculate global position
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x >= w-1 || y >= h-1 || x < 1 || y < 1)
        return;
    
    // Compute pixel index - ensure coalesced memory access by accessing
    // sequential memory locations within a warp
    const int idx = (y * w + x) * channels;
    
    // Process each pixel - optimized to reduce repeated calculations
    unsigned char r = d_img[idx];
    unsigned char g = d_img[idx+1];
    unsigned char b = d_img[idx+2];
    const unsigned char gray = (r + g + b) / 3;
    
    // Pre-compute indices for neighboring pixels to improve memory access patterns
    const int idx_left = (y * w + (x-1)) * channels;
    const int idx_right = (y * w + (x+1)) * channels;
    const int idx_up = ((y-1) * w + x) * channels;
    const int idx_down = ((y+1) * w + x) * channels;
    
    // Calculate grayscale values with fewer arithmetic operations
    const unsigned char gray_left = (d_img[idx_left] + d_img[idx_left+1] + d_img[idx_left+2]) / 3;
    const unsigned char gray_right = (d_img[idx_right] + d_img[idx_right+1] + d_img[idx_right+2]) / 3;
    const unsigned char gray_up = (d_img[idx_up] + d_img[idx_up+1] + d_img[idx_up+2]) / 3;
    const unsigned char gray_down = (d_img[idx_down] + d_img[idx_down+1] + d_img[idx_down+2]) / 3;
    
    // Calculate gradients
    const int h_gradient = abs(gray_right - gray_left);
    const int v_gradient = abs(gray_down - gray_up);
    
    // Detect edges using constant memory threshold
    if (h_gradient > c_edge_threshold || v_gradient > c_edge_threshold) {
        local.edge_pixels++;
        
        // Use direct assignment instead of atomic add for thread-local values
        // Will be accumulated to shared memory later
        // Check for regular edges (straight lines common in UI)
        if ((h_gradient > c_edge_threshold && v_gradient < c_edge_threshold/2) || 
            (v_gradient > c_edge_threshold && h_gradient < c_edge_threshold/2)) {
            local.regular_edge_pixels++;
        }
        
        // Atomic update to row-specific counter - cannot be easily avoided
        atomicAdd(&d_horizontal_edge_counts[y], 1);
    }
    
    // Check for uniform color regions with optimized memory access
    int local_variance = 0;
    for (int dy = -1; dy <= 1; dy++) {
        for (int dx = -1; dx <= 1; dx++) {
            if (y+dy >= 0 && y+dy < h && x+dx >= 0 && x+dx < w) {
                const int local_idx = ((y+dy) * w + (x+dx)) * channels;
                const unsigned char local_gray = (d_img[local_idx] + d_img[local_idx+1] + d_img[local_idx+2]) / 3;
                local_variance += abs(gray - local_gray);
            }
        }
    }
    
    // Low local variance indicates uniform color region
    if (local_variance < 20) {
        local.uniform_color_pixels++;
    }
    
    // Accumulate local counters to shared memory - reduced atomic operations
    atomicAdd(&s_edge_pixels, local.edge_pixels);
    atomicAdd(&s_regular_edge_pixels, local.regular_edge_pixels);
    atomicAdd(&s_uniform_color_pixels, local.uniform_color_pixels);
    
    // Ensure all threads complete before final accumulation
    __syncthreads();
    
    // Only one thread per block updates the global counters
    if (threadIdx.x == 0 && threadIdx.y == 0) {
        atomicAdd(d_edge_pixels, s_edge_pixels);
        atomicAdd(d_regular_edge_pixels, s_regular_edge_pixels);
        atomicAdd(d_uniform_color_pixels, s_uniform_color_pixels);
    }
}

// CUDA kernel for analyzing horizontal alignments with optimized memory access
__global__ void analyzeGridAlignmentKernel(
    const int* d_horizontal_edge_counts,
    int h, int w,
    int* d_aligned_rows) {
    
    // Use shared memory to cache edge counts for a block of rows
    extern __shared__ int s_edge_counts[];
    
    int y = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Load edge counts into shared memory for this thread block
    if (y < h) {
        s_edge_counts[threadIdx.x] = d_horizontal_edge_counts[y];
    }
    
    __syncthreads();
    
    // Each thread handles one row and checks alignment with next row
    if (y >= h-3 || y < 1)
        return;
    
    // Use shared memory for faster access when possible
    int curr_count = s_edge_counts[threadIdx.x];
    
    // For the next row, check if it's within the same thread block
    int next_row_idx = threadIdx.x + 1;
    int next_count;
    
    if (next_row_idx < blockDim.x && y + 1 < h) {
        // Next row is in shared memory
        next_count = s_edge_counts[next_row_idx];
    } else {
        // Next row is outside this block, read from global memory
        next_count = d_horizontal_edge_counts[y + 1];
    }
    
    // Check for similar edge patterns in consecutive rows
    if (curr_count > 0 && abs(curr_count - next_count) < w * 0.05) {
        atomicAdd(d_aligned_rows, 1);
    }
}

// Compute screenshot statistics with CUDA
ScreenshotStats computeScreenshotStatisticsGPU(unsigned char *img, int w, int h, int channels) {
    ScreenshotStats stats = {0};
    int total_pixels = w * h;
    
    // Allocate device memory
    unsigned char* d_img;
    int* d_edge_pixels;
    int* d_regular_edge_pixels;
    int* d_uniform_color_pixels;
    int* d_horizontal_edge_counts;
    int* d_aligned_rows;
    
    CUDA_CHECK(hipMalloc(&d_img, w * h * channels * sizeof(unsigned char)));
    CUDA_CHECK(hipMalloc(&d_edge_pixels, sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_regular_edge_pixels, sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_uniform_color_pixels, sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_horizontal_edge_counts, h * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_aligned_rows, sizeof(int)));
    
    // Initialize counters to 0
    CUDA_CHECK(hipMemset(d_edge_pixels, 0, sizeof(int)));
    CUDA_CHECK(hipMemset(d_regular_edge_pixels, 0, sizeof(int)));
    CUDA_CHECK(hipMemset(d_uniform_color_pixels, 0, sizeof(int)));
    CUDA_CHECK(hipMemset(d_horizontal_edge_counts, 0, h * sizeof(int)));
    CUDA_CHECK(hipMemset(d_aligned_rows, 0, sizeof(int)));
    
    // Copy image to device - ensure proper alignment for best performance
    CUDA_CHECK(hipMemcpy(d_img, img, w * h * channels * sizeof(unsigned char), hipMemcpyHostToDevice));
    
    // Launch kernels with optimized block size for better occupancy
    // Use 16x16 thread blocks for 2D data processing (good for coalescing memory access)
    dim3 blockSize(16, 16);
    dim3 gridSize((w + blockSize.x - 1) / blockSize.x, (h + blockSize.y - 1) / blockSize.y);
    
    computeScreenshotStatsKernel<<<gridSize, blockSize>>>(
        d_img, w, h, channels,
        d_edge_pixels, d_regular_edge_pixels, d_uniform_color_pixels,
        d_horizontal_edge_counts
    );
    CUDA_CHECK_KERNEL();
    
    // Launch grid alignment kernel with shared memory
    int blockSizeAlign = 256; // Optimize for occupancy
    int gridSizeAlign = (h + blockSizeAlign - 1) / blockSizeAlign;
    int sharedMemSize = blockSizeAlign * sizeof(int); // Allocate shared memory for edge counts
    
    analyzeGridAlignmentKernel<<<gridSizeAlign, blockSizeAlign, sharedMemSize>>>(
        d_horizontal_edge_counts, h, w, d_aligned_rows
    );
    CUDA_CHECK_KERNEL();
    
    // Copy results back to host
    int edge_pixels = 0, regular_edge_pixels = 0, uniform_color_pixels = 0, aligned_rows = 0;
    CUDA_CHECK(hipMemcpy(&edge_pixels, d_edge_pixels, sizeof(int), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(&regular_edge_pixels, d_regular_edge_pixels, sizeof(int), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(&uniform_color_pixels, d_uniform_color_pixels, sizeof(int), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(&aligned_rows, d_aligned_rows, sizeof(int), hipMemcpyDeviceToHost));
    
    // Calculate final statistics (normalized to [0,1] range)
    float edge_density = (float)edge_pixels / total_pixels;
    float edge_regularity = edge_pixels > 0 ? (float)regular_edge_pixels / edge_pixels : 0;
    float grid_alignment = (float)aligned_rows / h;
    float color_uniformity = (float)uniform_color_pixels / total_pixels;
    
    // Combine metrics into simplified scores
    stats.edge_score = (edge_regularity * 0.6) + (edge_density * 0.2) + (grid_alignment * 0.2);
    stats.color_score = color_uniformity;
    stats.ui_element_score = edge_density * 0.5 + grid_alignment * 0.5;
    
    // Cleanup
    CUDA_CHECK(hipFree(d_img));
    CUDA_CHECK(hipFree(d_edge_pixels));
    CUDA_CHECK(hipFree(d_regular_edge_pixels));
    CUDA_CHECK(hipFree(d_uniform_color_pixels));
    CUDA_CHECK(hipFree(d_horizontal_edge_counts));
    CUDA_CHECK(hipFree(d_aligned_rows));
    
    return stats;
}

// Load model from file
Feature* loadModel(const char* filename, int* size) {
    FILE* f = fopen(filename, "rb");
    if (!f) {
        fprintf(stderr, "Failed to open model file: %s\n", filename);
        return NULL;
    }
    
    // Read dataset size
    if (fread(size, sizeof(int), 1, f) != 1) {
        fprintf(stderr, "Failed to read model size\n");
        fclose(f);
        return NULL;
    }
    
    // Allocate memory for features
    Feature* model = (Feature*)malloc(*size * sizeof(Feature));
    if (!model) {
        fprintf(stderr, "Memory allocation failed\n");
        fclose(f);
        return NULL;
    }
    
    // Read features
    if (fread(model, sizeof(Feature), *size, f) != *size) {
        fprintf(stderr, "Failed to read model data\n");
        free(model);
        fclose(f);
        return NULL;
    }
    
    fclose(f);
    return model;
}

// Print device information
void printDeviceInfo() {
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDeviceProperties(&prop, 0));
    
    printf("\nCUDA Device Information:\n");
    printf("------------------------\n");
    printf("Device Name: %s\n", prop.name);
    printf("Compute Capability: %d.%d\n", prop.major, prop.minor);
    printf("Total Global Memory: %.2f GB\n", 
           (float)prop.totalGlobalMem / (1024.0f * 1024.0f * 1024.0f));
    printf("\n");
}

int main(int argc, char** argv) {
    if (argc != 3) {
        fprintf(stderr, "Usage: %s <model_file> <image_path>\n", argv[0]);
        return 1;
    }
    
    // Parse command line arguments
    const char* model_path = argv[1];
    const char* image_path = argv[2];
    
    // Performance timing
    clock_t start_time, end_time;
    double load_model_time = 0.0, feature_time = 0.0, classification_time = 0.0;
    
    // Initialize CUDA
    CUDA_CHECK(hipSetDevice(0));
    printDeviceInfo();
    
    // Load model
    start_time = clock();
    int model_size;
    Feature* model = loadModel(model_path, &model_size);
    end_time = clock();
    load_model_time = (double)(end_time - start_time) / CLOCKS_PER_SEC;
    
    if (!model) {
        return 1;
    }
    
    printf("Model loaded with %d training examples\n", model_size);
    
    // Load query image
    int width, height, channels;
    unsigned char* img = stbi_load(image_path, &width, &height, &channels, 3);
    if (!img) {
        fprintf(stderr, "Failed to load image: %s\n", image_path);
        free(model);
        return 1;
    }
    
    printf("Image loaded: %dx%d with %d channels\n", width, height, channels);
    
    // Check with statistical analysis first
    start_time = clock();
    ScreenshotStats stats = computeScreenshotStatisticsGPU(img, width, height, 3);
    int statistical_detection = isLikelyScreenshot(stats);
    
    // Extract features
    Feature query_feature;
    extractFeaturesGPU(img, 1, width, height, 3, &query_feature);
    end_time = clock();
    feature_time = (double)(end_time - start_time) / CLOCKS_PER_SEC;
    
    // If statistical detection is positive, skip kNN
    if (statistical_detection) {
        printf("Classification result for %s: SCREENSHOT (Statistical analysis)\n", image_path);
        printf("This image was detected as a screenshot by analyzing UI patterns\n");
        query_feature.label = 2; // Mark as detected by statistical analysis
    } else {
        // Perform kNN classification on GPU
        start_time = clock();
        int prediction;
        double knn_times[2] = {0}; // [0] = transfer time, [1] = compute time
        
        // Call GPU KNN function (we only have 1 query)
        classifyBatchGPU(model, model_size, &query_feature, 1, &prediction, knn_times);
        
        end_time = clock();
        classification_time = knn_times[0] + knn_times[1];
        
        printf("Classification result for %s: %s\n", image_path, 
               prediction ? "SCREENSHOT" : "NON-SCREENSHOT");
        printf("Classification based on K-nearest neighbors (K=%d)\n", K_NEIGHBORS);
    }
    
    // Print performance metrics
    printf("\nPerformance Metrics:\n");
    printf("-------------------\n");
    printf("Model Loading Time: %.5f seconds\n", load_model_time);
    printf("Feature Extraction Time: %.5f seconds\n", feature_time);
    printf("Classification Time: %.5f seconds\n", classification_time);
    printf("Total Processing Time: %.5f seconds\n", 
           load_model_time + feature_time + classification_time);
    printf("Model Memory Usage: %.2f MB\n", 
           (float)(model_size * sizeof(Feature)) / (1024.0f * 1024.0f));
    
    // Clean up
    stbi_image_free(img);
    free(model);
    
    return 0;
} 